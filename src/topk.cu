#include "hip/hip_runtime.h"
#include "topk.h"

typedef uint4 group_t;  // uint32_t
const size_t group_size = 8;

void __global__ docQueryScoringCoalescedMemoryAccessSampleKernel(
    const __restrict__ uint16_t *docs, const int *doc_lens, const size_t n_docs,
    uint16_t *query, const int query_len, float *scores) {
  // each thread process one doc-query pair scoring task
  register auto tid = blockIdx.x * blockDim.x + threadIdx.x,
                tnum = gridDim.x * blockDim.x;

  if (tid >= n_docs) {
    return;
  }

  __shared__ uint16_t query_on_shm[MAX_QUERY_SIZE];
#pragma unroll
  for (auto i = threadIdx.x; i < query_len; i += blockDim.x) {
    query_on_shm[i] = query[i];  // 不太高效的查询加载，假设它不是热点
  }

  __syncthreads();

  for (auto doc_id = tid; doc_id < n_docs; doc_id += tnum) {
    register int query_idx = 0;

    register float tmp_score = 0.;

    register bool no_more_load = false;

    for (auto i = 0; i < MAX_DOC_SIZE / group_size; i++) {
      if (no_more_load) {
        break;
      }
      register group_t loaded = ((group_t *)docs)[i * n_docs + doc_id];  // tid
      register uint16_t *doc_segment = (uint16_t *)(&loaded);
      for (auto j = 0; j < group_size; j++) {
        if (doc_segment[j] == 0) {
          no_more_load = true;
          break;
          // return;
        }
        int left = query_idx;
        int right = query_len - 1;
        int mid;
        while (left <= right) {
          mid = (left + right) / 2;
          if (query_on_shm[mid] < doc_segment[j]) {
            left = mid + 1;
          } else {
            right = mid - 1;
          }
        }
        query_idx = left;  // update the query index

        if (query_idx < query_len) {
          tmp_score += (query_on_shm[query_idx] == doc_segment[j]);
        }
      }
      __syncwarp();
    }
    scores[doc_id] = tmp_score / max(query_len, doc_lens[doc_id]);  // tid
  }
}

void doc_query_scoring_gpu_function(
    std::vector<std::vector<uint16_t>> &querys,
    std::vector<std::vector<uint16_t>> &docs, std::vector<uint16_t> &lens,
    std::vector<std::vector<int>> &indices, uint16_t *h_docs,
    std::vector<int> &h_doc_lens_vec  // shape [querys.size(), TOPK]
) {
  // printf("uint4 %lu group_t %lu uint16_t %lu rest %lu
  // \n",sizeof(uint4),sizeof(group_t),sizeof(uint16_t),sizeof(group_t) /
  // sizeof(uint16_t));
  auto n_docs = docs.size();
  std::vector<float> scores(n_docs);
  std::vector<int> s_indices(n_docs);
  float *d_scores = nullptr;
  uint16_t *d_docs = nullptr, *d_query = nullptr;
  int *d_doc_lens = nullptr;

  hipStream_t stream = hipStreamPerThread;
  // copy to device
  hipMallocAsync(&d_docs, sizeof(uint16_t) * MAX_DOC_SIZE * n_docs, stream);
  hipMallocAsync(&d_scores, sizeof(float) * n_docs, stream);
  hipMallocAsync(&d_doc_lens, sizeof(int) * n_docs, stream);

  hipMemcpyAsync(d_docs, h_docs, sizeof(uint16_t) * MAX_DOC_SIZE * n_docs,
                  hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(d_doc_lens, h_doc_lens_vec.data(), sizeof(int) * n_docs,
                  hipMemcpyHostToDevice, stream);

  hipDeviceProp_t device_props;
  hipGetDeviceProperties(&device_props, 0);

  hipSetDevice(0);

  int block = N_THREADS_IN_ONE_BLOCK;
  int grid = (n_docs + block - 1) / block;
#pragma unroll
  for (int i = 0; i < n_docs; ++i) {
    s_indices[i] = i;
  }
  int index = 0;
  for (auto &query : querys) {
    // init indices

    const size_t query_len = query.size();
    hipMallocAsync(&d_query, sizeof(uint16_t) * query_len, stream);
    hipMemcpyAsync(d_query, query.data(), sizeof(uint16_t) * query_len,
                    hipMemcpyHostToDevice, stream);
    // launch kernel

    docQueryScoringCoalescedMemoryAccessSampleKernel<<<grid, block, 0,
                                                       stream>>>(
        d_docs, d_doc_lens, n_docs, d_query, query_len, d_scores);

    if (index++ != 0) {
      std::partial_sort(s_indices.begin(), s_indices.begin() + TOPK,
                        s_indices.end(), [&scores](const int &a, const int &b) {
                          if (scores[a] != scores[b]) {
                            return scores[a] > scores[b];  // 按照分数降序排序
                          }
                          return a < b;  // 如果分数相同，按索引从小到大排序
                        });
      std::vector<int> s_ans(s_indices.begin(), s_indices.begin() + TOPK);
      indices.push_back(s_ans);
    }

    hipMemcpyAsync(scores.data(), d_scores, sizeof(float) * n_docs,
                    hipMemcpyDeviceToHost, stream);
  }

  // hipStreamSynchronize(stream);
  std::partial_sort(s_indices.begin(), s_indices.begin() + TOPK,
                    s_indices.end(), [&scores](const int &a, const int &b) {
                      if (scores[a] != scores[b]) {
                        return scores[a] > scores[b];  // 按照分数降序排序
                      }
                      return a < b;  // 如果分数相同，按索引从小到大排序
                    });
  std::vector<int> s_ans(s_indices.begin(), s_indices.begin() + TOPK);
  indices.push_back(s_ans);

  hipFreeAsync(d_query, stream);

  // deallocation
  hipFree(d_docs);
  // hipFreeAsync(d_query);
  hipFree(d_scores);
  hipFree(d_doc_lens);
  // free(h_docs);
}
