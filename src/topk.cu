#include "hip/hip_runtime.h"
#include <nvtx3/nvToolsExt.h>
#include <hipcub/hipcub.hpp>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>

// #include <numeric>
#include <thrust/execution_policy.h>
#include <thrust/scan.h>

#include <numeric>
#include <thread>

#include "assert.h"
#include "topk.h"

typedef uint4 group_t;  // uint32_t
#define CHECK(res)          \
  if (res != hipSuccess) { \
    exit(-1);               \
  }
#define GROUP_SIZE 8

dim3 numBlocks(32, 32);
dim3 threadsPerBlock(32, 32);

void __global__ docQueryScoringCoalescedMemoryAccessSampleKernel(
    const __restrict__ uint16_t *docs, const uint16_t *doc_lens,
    const size_t n_docs, const uint16_t *query, const int query_len,
    float *scores, int *d_index, size_t doc_size) {
  // each thread process one doc-query pair scoring task
  register auto tid = blockIdx.x * blockDim.x + threadIdx.x,
                tnum = gridDim.x * blockDim.x;

  if (tid >= n_docs) {
    return;
  }

  __shared__ uint32_t query_on_shm[MAX_QUERY_SIZE];

#pragma unroll
  for (auto i = threadIdx.x; i < query_len; i += blockDim.x) {
    query_on_shm[i] = query[i];  // 不太高效的查询加载，假设它不是热点
  }

  __syncthreads();

  for (auto doc_id = tid; doc_id < n_docs; doc_id += tnum) {
    register int query_idx = 0;

    register float tmp_score = 0.;

    register bool no_more_load = false;

    register size_t doc_len = n_docs >> 3;

    for (auto i = 0; i < doc_len; i++) {
      if (no_more_load) {
        break;
      }
      register group_t loaded = ((group_t *)docs)[i * n_docs + doc_id];  // tid
      register uint16_t *doc_segment = (uint16_t *)(&loaded);
      for (auto j = 0; j < GROUP_SIZE; j++) {
        if (doc_segment[j] == 0) {
          no_more_load = true;
          break;
          // return;
        }
        int left = query_idx;
        int right = query_len - 1;
        int mid;
        while (left <= right) {
          mid = (left + right) >> 1;
          if (query_on_shm[mid] < doc_segment[j]) {
            left = mid + 1;
          } else {
            right = mid - 1;
          }
        }
        query_idx = left;  // update the query index

        if (query_idx < query_len) {
          tmp_score += (query_on_shm[query_idx] == doc_segment[j]);
        }
      }
      __syncwarp();
    }
    scores[doc_id] = tmp_score / max(query_len, doc_lens[doc_id]);  // tid
    d_index[doc_id] = doc_id;
  }
}
//  TODO  理清楚uint4 uin2 取址 解决255报错 然后保证性能 难道32同步很难? 
__global__ void countMatchesOptimized(
            const __restrict__  uint16_t* docs,
            const uint16_t *doc_lens,
            const uint16_t* query, 
            float* scores, 
            const size_t n_docs, 
            const int query_len,
            int *d_index) {
    register auto doc_id = blockIdx.x * blockDim.x  + threadIdx.x ;
    register auto  tnumx = blockDim.x * gridDim.x ;
    register auto  lane_id = threadIdx.x & 0x1f;
    register auto warp_id = threadIdx.x >> 5;
    register int aggregate = 0;
    register uint16_t match_count = 0;
    register bool match = false;
    register uint16_t test = warp_id;
    register uint16_t  query_val  = 0;
    doc_id = doc_id >>5;
    tnumx = tnumx >>5;
    typedef hipcub::WarpReduce<int> WarpReduce;

    // __shared__ uint32_t shared_query[ 32][129]; // 假设 query 长度最大为 128
    __shared__ uint32_t shared_query[129]; // 假设 query 长度最大为 128

    __shared__ typename WarpReduce::TempStorage temp_storage[N_THREADS_IN_ONE_BLOCK / 32];   // 512/32 -> 16

    if (threadIdx.x < query_len) {
        // #pragma unroll
        // register uint32_t query_tmpt =  query[threadIdx.x];
        // for(auto j = 0; j < 32 ; j++)
        // {
        //     shared_query[j][threadIdx.x] = query_tmpt;
        // }

            shared_query[threadIdx.x] = query[threadIdx.x];

    }
    __syncthreads();
// 现在query 所在的共享内存读取还是太多 看看能不能就是只读query len次数/. doc 过一次 query  n_doc 次数
// 耗时过高 可能的原因 寄存器内存花费过多 
// 共享内存耗时高

// 好像这样会因小失大 造成doc的巨量读取  doc query_len次数 query 1次
    #pragma unroll
    for (auto doc_index = doc_id; doc_index < n_docs; doc_index += tnumx) {
        // 这里没对齐访问? 
        register uint2 doc_data = ((uint2 *)docs)[doc_index * 32 + lane_id];


        match_count = 0;
        // aggregate = 0;
        #pragma unroll
        for (int i = 0; i < query_len; i++) {
            // if (lane_id == 0) query_val= shared_query[lane_id][i];  // 光share的访问就到1ms?  说好的广播呢 呜呜呜。
            query_val= shared_query[i] +1 ;  // 光share的访问就到1ms?  说好的广播呢 呜呜呜。

            // 找高效的宏 好像是这里耗时太多了 
            match = false;
            // match |= ( ((uint16_t *)(&doc_data))[0]== query_val);
            // match |= ( ((uint16_t *)(&doc_data))[1]== query_val);
            // match |= ( ((uint16_t *)(&doc_data))[2]== query_val);
            // match |= ( ((uint16_t *)(&doc_data))[3]== query_val);
            match |= ( 12== query_val);
            match |= ( 123== query_val);
            match |= ( 124== query_val);
            match |= ( 125== query_val);

            match_count +=match;

        }
        // __syncwarp();
        // __syncthreads();

        aggregate = WarpReduce(temp_storage[warp_id]).Sum(match_count);  // warp加了后 耗时从700us -> 1600us (线程同步) -> 14000 warp 同步



        if (lane_id == 0) {
            scores[doc_index] =aggregate;/// max(doc_lens[doc_index], query_len);
            d_index[doc_index] = doc_index;
        }
        // __syncwarp();
    }

}
__global__ void pre_process_global_no_reshape_shared(const uint16_t *temp_docs, uint16_t *d_docs,
    const uint16_t *d_doc_lens,
    const size_t n_docs,
    const uint32_t *d_doc_sum) {
    // 定义共享内存
    // __shared__ uint16_t shared_data[128]; // 假设每个线程块处理的最大文档长度为 128

    register auto tidx = blockIdx.x * blockDim.x + threadIdx.x;
    register auto doc_index = tidx /128;
    register auto stride = (blockDim.x  * gridDim.x ) / 128;
    register auto doc_lane = threadIdx.x % 128;

    for (auto doc_id =  doc_index; doc_id< n_docs; doc_id +=stride) {
        register auto offset = doc_id * 128;
        register auto d_lens = d_doc_lens[tidx];
        d_docs[offset + doc_lane]  =  doc_lane < d_doc_lens[doc_id] ? temp_docs [d_doc_sum[doc_id] + doc_lane]: 0;
    }
}

void pre_process(std::vector<std::vector<uint16_t>> &docs, uint16_t *h_docs,
                 uint32_t *h_docs_vec, size_t start_idx, size_t lens) {
// h_docs_vec[0] = 0;
#pragma unroll
  for (size_t i = start_idx; i < lens; i++) {
    auto doc_size = docs[i].size();
    memcpy(h_docs + h_docs_vec[i], &docs[i][0], doc_size * sizeof(uint16_t));
  }
}

void prepare_1(uint32_t **h_docs_vec, std::vector<uint16_t> &lens,
               size_t *doc_size, size_t n_docs) {
  auto it = max_element(std::begin(lens), std::end(lens));
  *doc_size = (*it + 8 >> 3) << 3;
  *h_docs_vec = new uint32_t[n_docs + 1];
  std::copy(lens.begin(), lens.end(), *h_docs_vec + 1);
  std::partial_sum(*h_docs_vec + 1, *h_docs_vec + n_docs + 1, *h_docs_vec + 1);
}
void prepare_2(std::vector<std::vector<uint16_t>> &querys,
               uint16_t *max_query) {
  for (size_t i = 0; i < querys.size(); i++) {
    auto it = max_element(std::begin(querys[i]), std::end(querys[i]));
    *max_query = max((*it), *max_query);
  }
}

void d_docs_malloc(uint16_t **d_docs, size_t n_docs, size_t doc_size) {
  // hipSetDevice(0);
  hipMalloc(d_docs, sizeof(uint16_t) * doc_size * n_docs);
}
void d_sort_scores_malloc(float **d_sort_scores, int **s_indices,
                          size_t n_docs) {
  // hipSetDevice(0);
  hipMalloc(d_sort_scores, sizeof(float) * n_docs);
  CHECK(hipMalloc(s_indices, sizeof(int) * n_docs));
}
void d_sort_index_malloc(int **d_sort_index, float **d_scores, size_t n_docs) {
  // hipSetDevice(0);
  CHECK(hipMalloc(d_scores, sizeof(float) * n_docs));
  hipMalloc(d_sort_index, sizeof(int) * n_docs);
}

void temp_docs_copy(uint16_t **temp_docs, uint16_t *h_docs,
                    uint32_t *h_docs_vec, size_t n_docs) {
  // hipSetDevice(0);
  hipMalloc(temp_docs, sizeof(uint16_t) * h_docs_vec[n_docs]);
  CHECK(hipMemcpy(*temp_docs, h_docs, sizeof(uint16_t) * h_docs_vec[n_docs],
                   hipMemcpyHostToDevice));
}

void d_doc_lens_malloc(uint16_t **d_doc_lens, std::vector<uint16_t> &lens,
                       size_t n_docs) {
  // hipSetDevice(0);
  hipMalloc(d_doc_lens, sizeof(uint16_t) * n_docs);
  CHECK(hipMemcpy(*d_doc_lens, lens.data(), sizeof(uint16_t) * n_docs,
                   hipMemcpyHostToDevice));
}
void d_doc_sum_copy(uint32_t **d_doc_sum, uint16_t **temp_docs,
                    uint32_t *h_docs_vec, std::vector<uint16_t> &lens,
                    size_t n_docs) {
  // hipSetDevice(0);
  hipMalloc(d_doc_sum, sizeof(uint32_t) * (n_docs + 1));

  CHECK(hipMemcpy(*d_doc_sum, h_docs_vec, sizeof(uint32_t) * (n_docs + 1),
                   hipMemcpyHostToDevice));
}

int block = N_THREADS_IN_ONE_BLOCK;

void doc_query_scoring_gpu_function(
    std::vector<std::vector<uint16_t>> &querys,
    std::vector<std::vector<uint16_t>> &docs, std::vector<uint16_t> &lens,
    std::vector<std::vector<int>> &indices  // shape [querys.size(), TOPK]
) {
  // std::chrono::high_resolution_clock::time_point t1 =
  //     std::chrono::high_resolution_clock::now();

  register size_t n_docs = docs.size();
  int grid = (n_docs + block - 1) / block;
  int querys_len = querys.size();

  int *d_sort_index = nullptr;
  float *d_sort_scores = nullptr;
  void *d_temp_storage = nullptr;
  uint32_t *d_doc_sum = nullptr;
  uint16_t *temp_docs = nullptr;
  size_t temp_storage_bytes = 0;

  uint16_t *d_docs = nullptr;
  uint16_t *d_doc_lens = nullptr;

  uint32_t *h_docs_vec = nullptr;
  size_t doc_size = 0;
  uint16_t max_query = 0;

  float *d_scores = nullptr;
  int *s_indices = nullptr;
  uint16_t *d_query = nullptr;
  hipStream_t *streams;

  std::thread prepare_thread_1(prepare_1, &h_docs_vec, std::ref(lens),
                               &doc_size, n_docs);
  std::thread prepare_thread_2(prepare_2, std::ref(querys), &max_query);

  std::thread malloc_thread_2(d_sort_scores_malloc, &d_sort_scores, &s_indices,
                              n_docs);
  std::thread malloc_thread_3(d_sort_index_malloc, &d_sort_index, &d_scores,
                              n_docs);
  std::thread malloc_thread_4(d_doc_lens_malloc, &d_doc_lens, std::ref(lens),
                              n_docs);
  prepare_thread_1.join();
  std::thread malloc_thread_1(d_docs_malloc, &d_docs, n_docs, doc_size);
  std::thread malloc_thread_5(d_doc_sum_copy, &d_doc_sum, &temp_docs,
                              h_docs_vec, std::ref(lens), n_docs);

  uint16_t *h_docs = new uint16_t[doc_size * n_docs];
  size_t num_threads = 10;
  std::vector<std::thread> threads(num_threads);
  std::vector<std::thread> s_threads(querys_len);

  register size_t chunk_size = n_docs / num_threads;  // 分块大小
  for (size_t i = 0; i < num_threads; i++) {
    size_t start = i * chunk_size;
    size_t end = (i == num_threads - 1) ? n_docs : start + chunk_size;
    threads[i] = std::thread(pre_process, std::ref(docs), h_docs, h_docs_vec,
                             start, end);
  }

  std::vector<std::vector<int>> indices_pre(querys_len, std::vector<int>(TOPK));

  for (std::thread &t : threads) {
    t.join();  // 等待所有线程完成
  }

  std::thread copy_thread_1(temp_docs_copy, &temp_docs, h_docs, h_docs_vec,
                            n_docs);

  streams = (hipStream_t *)malloc(querys_len * sizeof(hipStream_t));
  for (int i = 0; i < querys_len; ++i) {
    CHECK(hipStreamCreate(&streams[i]));
  }

  prepare_thread_2.join();
  CHECK(hipMallocAsync(&d_query, sizeof(uint16_t) * max_query, streams[0]));
  // std::chrono::high_resolution_clock::time_point t4 =
  //     std::chrono::high_resolution_clock::now();
  malloc_thread_2.join();
  malloc_thread_3.join();
  hipcub::DeviceRadixSort::SortPairsDescending(d_temp_storage, temp_storage_bytes,
                                            d_scores, d_sort_scores, s_indices,
                                            d_sort_index, n_docs);
  CHECK(hipMallocAsync(&d_temp_storage, temp_storage_bytes, streams[0]));

  malloc_thread_1.join();
  malloc_thread_4.join();
  malloc_thread_5.join();
  copy_thread_1.join();

  // std::chrono::high_resolution_clock::time_point t5 =
  //     std::chrono::high_resolution_clock::now();
  // std::cout
  //     << "first join cost "
  //     << std::chrono::duration_cast<std::chrono::milliseconds>(t5 -
  //     t4).count()
  //     << " ms " << std::endl;

  // std::chrono::high_resolution_clock::time_point t5 =
  //     std::chrono::high_resolution_clock::now();
  // std::cout
  //     << "before pre_process_global cost "
  //     << std::chrono::duration_cast<std::chrono::milliseconds>(t5 -
  //     t1).count()
  //     << " ms " << std::endl;
  pre_process_global_no_reshape_shared<<<grid, block>>>(
      temp_docs, d_docs, d_doc_lens, n_docs, d_doc_sum);


  // std::chrono::high_resolution_clock::time_point t6 =
  //     std::chrono::high_resolution_clock::now();
  // std::cout
  //     << "init cost "
  //     << std::chrono::duration_cast<std::chrono::milliseconds>(t6 -
  //     t1).count()
  //     << " ms " << std::endl;

  std::cout
        << "uint4 "
        << sizeof(uint4)    //16 
        << "uint2 "         //8
        << sizeof(uint2)
        << "uint16_t "      //2
        << sizeof(uint16_t)
        << " ms " << std::endl;
        
  for (int i = 0; i < querys_len; ++i) {
    auto &query = querys[i];
    const size_t query_len = query.size();
    // nvtxRangePushA("cuda malloc");
    CHECK(hipMemcpyAsync(d_query, query.data(), sizeof(uint16_t) * query_len,
                          hipMemcpyHostToDevice, streams[i]));

    // docQueryScoringCoalescedMemoryAccessSampleKernel<<<grid, block, 0,
    //                                                    streams[i]>>>(
    //     d_docs, d_doc_lens, n_docs, d_query, query_len, d_scores, s_indices,
    //     doc_size);


    countMatchesOptimized<<<grid, block, 0,streams[i]>>>(
         d_docs,
         d_doc_lens, 
         d_query,
         d_scores,
         n_docs,
         query_len,
         s_indices);
    hipcub::DeviceRadixSort::SortPairsDescending(
        d_temp_storage, temp_storage_bytes, d_scores, d_sort_scores, s_indices,
        d_sort_index, n_docs);
    // nvtxRangePop();
    CHECK(hipMemcpyAsync(indices_pre[i].data(), d_sort_index,
                          sizeof(int) * TOPK, hipMemcpyDeviceToHost,
                          streams[i]));
  }
  indices = indices_pre;
  // deallocation
  CHECK(hipFreeAsync(d_scores, streams[0]));
  CHECK(hipFreeAsync(s_indices, streams[1]));
  CHECK(hipFreeAsync(d_query, streams[2]));
  CHECK(hipFreeAsync(d_temp_storage, streams[3]));
  CHECK(hipFreeAsync(d_docs, streams[4]));
  CHECK(hipFreeAsync(d_doc_lens, streams[5]));
  // free(h_docs);
}
