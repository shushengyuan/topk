#include "hip/hip_runtime.h"
#include <nvtx3/nvToolsExt.h>
// #include <omp.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>

// #include <numeric>
#include <thrust/execution_policy.h>
#include <thrust/scan.h>

#include <numeric>
#include <thread>

#include "assert.h"
#include "topk.h"
#include "unistd.h"

typedef uint4 group_t;  // uint32_t
#define CHECK(res)          \
  if (res != hipSuccess) { \
    exit(-1);               \
  }
#define GROUP_SIZE 8

void __global__ docQueryScoringCoalescedMemoryAccessSampleKernel(
    const __restrict__ uint16_t *docs, const uint16_t *doc_lens,
    const size_t n_docs, const uint16_t *query, const int query_len,
    float *scores, int *d_index) {
  // each thread process one doc-query pair scoring task
  register auto tid = blockIdx.x * blockDim.x + threadIdx.x,
                tnum = gridDim.x * blockDim.x;

  if (tid >= n_docs) {
    return;
  }

  __shared__ uint32_t query_on_shm[MAX_QUERY_SIZE];

#pragma unroll
  for (auto i = threadIdx.x; i < query_len; i += blockDim.x) {
    query_on_shm[i] = query[i];  // 不太高效的查询加载，假设它不是热点
  }

  __syncthreads();

  for (auto doc_id = tid; doc_id < n_docs; doc_id += tnum) {
    register int query_idx = 0;

    register float tmp_score = 0.;

    register bool no_more_load = false;

    register size_t doc_len = MAX_DOC_SIZE >> 3;

    for (auto i = 0; i < doc_len; i++) {
      if (no_more_load) {
        break;
      }
      register group_t loaded = ((group_t *)docs)[i * n_docs + doc_id];  // tid
      register uint16_t *doc_segment = (uint16_t *)(&loaded);
      for (auto j = 0; j < GROUP_SIZE; j++) {
        if (doc_segment[j] == 0) {
          no_more_load = true;
          break;
          // return;
        }
        int left = query_idx;
        int right = query_len - 1;
        int mid;
        while (left <= right) {
          mid = (left + right) >> 1;
          if (query_on_shm[mid] < doc_segment[j]) {
            left = mid + 1;
          } else {
            right = mid - 1;
          }
        }
        query_idx = left;  // update the query index

        if (query_idx < query_len) {
          tmp_score += (query_on_shm[query_idx] == doc_segment[j]);
        }
      }
      __syncwarp();
    }
    scores[doc_id] = tmp_score / max(query_len, doc_lens[doc_id]);  // tid
    d_index[doc_id] = doc_id;
  }
}
__global__ void pre_process_global(const uint16_t *temp_docs, uint16_t *d_docs,
                                   const uint16_t *d_doc_lens,
                                   const size_t n_docs,
                                   const uint32_t *d_doc_sum) {
  // register auto group_sz = 8;  // sizeof(group_t) / sizeof(uint16_t)
  register auto layer_0_stride = n_docs * 8;  // group_sz;
  // register auto layer_1_stride = 8;           // group_sz;

  register auto tidx = blockIdx.x * blockDim.x + threadIdx.x,
                tnumx = gridDim.x * blockDim.x;
  register auto tidy = blockIdx.y * blockDim.y + threadIdx.y,
                tnumy = gridDim.y * blockDim.y;
#pragma unroll
  for (auto i = tidx; i < n_docs; i += tnumx) {
    // register auto layer_1_offset = i;
    register auto layer_1_total_offset = i << 3;
    register auto base_id = d_doc_sum[i];
    register auto d_lens = d_doc_lens[i];
#pragma unroll
    for (auto j = tidy; j < d_lens; j += tnumy) {
      register auto layer_0_offset = j >> 3;  // group_sz;
      register auto layer_2_offset = j & 7;   // j % group_sz;
      register auto final_offset = layer_0_offset * layer_0_stride +
                                   layer_1_total_offset + layer_2_offset;
      d_docs[final_offset] = temp_docs[base_id + j];
    }
  }
}

void pre_process(std::vector<std::vector<uint16_t>> &docs, uint16_t *h_docs,
                 uint32_t *h_docs_vec, size_t start_idx, size_t lens) {
// h_docs_vec[0] = 0;
#pragma unroll
  for (size_t i = start_idx; i < lens; i++) {
    register auto doc_size = docs[i].size();
    // h_docs_vec[i + 1] = h_docs_vec[i] + doc_size;
    register int doc_id = h_docs_vec[i];

#pragma unroll
    for (size_t j = 0; j < doc_size; j++) {
      h_docs[doc_id + j] = docs[i][j];
    }
  }
}

void temp_docs_global_thread(uint16_t **temp_docs, uint16_t *h_docs,
                             uint32_t *h_docs_vec, size_t n_docs) {
  hipSetDevice(0);

  hipMalloc(temp_docs, sizeof(uint16_t) * h_docs_vec[n_docs]);
  CHECK(hipMemcpy(*temp_docs, h_docs, sizeof(uint16_t) * h_docs_vec[n_docs],
                   hipMemcpyHostToDevice));
}

void d_doc_sum_global_thread(uint32_t **d_doc_sum, uint32_t *h_docs_vec,
                             size_t n_docs) {
  hipSetDevice(0);
  hipMalloc(d_doc_sum, sizeof(uint32_t) * (n_docs + 1));
  CHECK(hipMemcpy(*d_doc_sum, h_docs_vec, sizeof(uint32_t) * (n_docs + 1),
                   hipMemcpyHostToDevice));
}

void malloc_global_thread_1(uint16_t **d_docs, size_t n_docs) {
  hipSetDevice(0);

  hipMalloc(d_docs, sizeof(uint16_t) * MAX_DOC_SIZE * n_docs);
}
void malloc_global_thread_2(int **d_sort_index, float **d_sort_scores,
                            uint16_t **d_doc_lens, std::vector<uint16_t> &lens,
                            size_t n_docs) {
  hipSetDevice(0);
  hipMalloc(d_sort_index, sizeof(int) * n_docs);
  hipMalloc(d_sort_scores, sizeof(float) * n_docs);
  hipMalloc(d_doc_lens, sizeof(uint16_t) * n_docs);
  CHECK(hipMemcpy(*d_doc_lens, lens.data(), sizeof(uint16_t) * n_docs,
                   hipMemcpyHostToDevice));
}

// void h_docs_vec_thread(uint16_t **h_docs, uint32_t **h_docs_vec,
//                        std::vector<uint16_t> &lens, size_t n_docs) {
//   *h_docs = new uint16_t[MAX_DOC_SIZE * n_docs];
//   *h_docs_vec = new uint32_t[n_docs + 1];
//   std::copy(lens.begin(), lens.end(), *h_docs_vec + 1);
//   std::partial_sum(*h_docs_vec + 1, *h_docs_vec + n_docs + 1, *h_docs_vec +
//   1);
// }

void doc_query_scoring_gpu_function(
    std::vector<std::vector<uint16_t>> &querys,
    std::vector<std::vector<uint16_t>> &docs, std::vector<uint16_t> &lens,
    std::vector<std::vector<int>> &indices  // shape [querys.size(), TOPK]
) {
  // std::chrono::high_resolution_clock::time_point t1 =
  //     std::chrono::high_resolution_clock::now();

  register size_t n_docs = docs.size();
  int block = N_THREADS_IN_ONE_BLOCK;
  int grid = (n_docs + block - 1) / block;
  int querys_len = querys.size();

  int *d_sort_index = nullptr;
  float *d_sort_scores = nullptr;
  void *d_temp_storage = nullptr;
  uint32_t *d_doc_sum = nullptr;
  uint16_t *temp_docs = nullptr;
  size_t temp_storage_bytes = 0;

  uint16_t *d_docs = nullptr;
  uint16_t *d_doc_lens = nullptr;

  hipDeviceProp_t device_props;
  hipGetDeviceProperties(&device_props, 0);
  hipSetDevice(0);

  hipStream_t *streams;
  // nvtxRangePushA("streams create");
  dim3 numBlocks(32, 32);
  dim3 threadsPerBlock(32, 32);

  std::thread malloc_thread_1(malloc_global_thread_1, &d_docs, n_docs);
  std::thread malloc_thread_2(malloc_global_thread_2, &d_sort_index,
                              &d_sort_scores, &d_doc_lens, std::ref(lens),
                              n_docs);

  // nvtxRangePushA("new *");

  uint16_t *h_docs = new uint16_t[MAX_DOC_SIZE * n_docs];
  uint32_t *h_docs_vec = new uint32_t[n_docs + 1];
  std::copy(lens.begin(), lens.end(), h_docs_vec + 1);
  std::partial_sum(h_docs_vec + 1, h_docs_vec + n_docs + 1, h_docs_vec + 1);

  std::thread copy_thread_2(d_doc_sum_global_thread, &d_doc_sum, h_docs_vec,
                            n_docs);

  size_t num_threads = 10;
  std::vector<std::thread> threads(num_threads);
  register size_t chunk_size = n_docs / num_threads;  // 分块大小
  for (size_t i = 0; i < num_threads; i++) {
    size_t start = i * chunk_size;
    size_t end = (i == num_threads - 1) ? n_docs : start + chunk_size;
    threads[i] = std::thread(pre_process, std::ref(docs), h_docs, h_docs_vec,
                             start, end);
  }

  for (std::thread &t : threads) {
    t.join();  // 等待所有线程完成
  }

  // std::thread copy_thread_3(temp_docs_global_thread, &temp_docs, h_docs,
  //                           h_docs_vec, n_docs);
  hipMalloc(&temp_docs, sizeof(uint16_t) * h_docs_vec[n_docs]);
  CHECK(hipMemcpy(temp_docs, h_docs, sizeof(uint16_t) * h_docs_vec[n_docs],
                   hipMemcpyHostToDevice));

  streams = (hipStream_t *)malloc(querys_len * sizeof(hipStream_t));
  std::vector<std::vector<int>> indices_pre(querys_len, std::vector<int>(TOPK));

  // copy_thread_3.join();
  copy_thread_2.join();
  malloc_thread_2.join();
  malloc_thread_1.join();

  pre_process_global<<<numBlocks, threadsPerBlock>>>(
      temp_docs, d_docs, d_doc_lens, n_docs, d_doc_sum);
  // std::chrono::high_resolution_clock::time_point t6 =
  //     std::chrono::high_resolution_clock::now();
  // std::cout
  //     << "init cost "
  //     << std::chrono::duration_cast<std::chrono::milliseconds>(t6 -
  //     t1).count()
  //     << " ms " << std::endl;

  for (int i = 0; i < querys_len; ++i) {
    // init indices
    // nvtxRangePushA("Loop start");
    CHECK(hipStreamCreate(&streams[i]));
    uint16_t *d_query = nullptr;
    float *d_scores = nullptr;
    int *s_indices = nullptr;

    auto &query = querys[i];
    const size_t query_len = query.size();
    // nvtxRangePushA("cuda malloc");
    CHECK(hipMallocAsync(&d_scores, sizeof(float) * n_docs, streams[i]));
    CHECK(hipMallocAsync(&s_indices, sizeof(int) * n_docs, streams[i]));
    CHECK(hipMallocAsync(&d_query, sizeof(uint16_t) * query_len, streams[i]));
    CHECK(hipMemcpyAsync(d_query, query.data(), sizeof(uint16_t) * query_len,
                          hipMemcpyHostToDevice, streams[i]));
    // nvtxRangePop();

    // nvtxRangePushA("topk kernal");
    docQueryScoringCoalescedMemoryAccessSampleKernel<<<grid, block, 0,
                                                       streams[i]>>>(
        d_docs, d_doc_lens, n_docs, d_query, query_len, d_scores, s_indices);
    // nvtxRangePop();

    // nvtxRangePushA("sort_by_key");
    if (i == 0) {
      hipcub::DeviceRadixSort::SortPairsDescending(
          d_temp_storage, temp_storage_bytes, d_scores, d_sort_scores,
          s_indices, d_sort_index, n_docs);
      // Allocate temporary storage
      CHECK(hipMallocAsync(&d_temp_storage, temp_storage_bytes, streams[i]));
    }
    hipcub::DeviceRadixSort::SortPairsDescending(
        d_temp_storage, temp_storage_bytes, d_scores, d_sort_scores, s_indices,
        d_sort_index, n_docs);
    // nvtxRangePop();
    CHECK(hipMemcpyAsync(indices_pre[i].data(), d_sort_index,
                          sizeof(int) * TOPK, hipMemcpyDeviceToHost,
                          streams[i]));
    CHECK(hipFreeAsync(s_indices, streams[i]));
    CHECK(hipFreeAsync(d_scores, streams[i]));
    CHECK(hipFreeAsync(d_query, streams[i]));
    // nvtxRangePop();
  }
  indices = indices_pre;
  // deallocation
  // hipFree(d_docs);
  // hipFree(d_doc_lens);
  // free(h_docs);
}