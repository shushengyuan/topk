#include "hip/hip_runtime.h"
#include <thread>
#include<omp.h>
#include <thrust/sort.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <nvtx3/nvToolsExt.h>

#include "topk.h"

typedef uint4 group_t;  // uint32_t
#define CHECK(res) if(res!=hipSuccess){exit(-1);}  

// device A100
//  cpu sort :
//  yuan trust sort L: 3002 ms
//  yuan trust sort L: 2750 ms



void __global__ docQueryScoringCoalescedMemoryAccessSampleKernel(
    const __restrict__ uint16_t *docs, const int *doc_lens, const size_t n_docs,
    uint16_t *query, const int query_len, float *scores, int *d_index) {
  // each thread process one doc-query pair scoring task
  register auto tid = blockIdx.x * blockDim.x + threadIdx.x,
                tnum = gridDim.x * blockDim.x;

  if (tid >= n_docs) {
    return;
  }

  __shared__ uint32_t query_on_shm[MAX_QUERY_SIZE];
  
#pragma unroll
  for (auto i = threadIdx.x; i < query_len; i += blockDim.x) {
    
    query_on_shm[i] = query[i];  // 不太高效的查询加载，假设它不是热点
  }

  __syncthreads();

  for (auto doc_id = tid; doc_id < n_docs; doc_id += tnum) {
    register int query_idx = 0;

    register float tmp_score = 0.;

    register bool no_more_load = false;

    for (auto i = 0; i < MAX_DOC_SIZE / GROUP_SIZE; i++) {
      if (no_more_load) {
        break;
      }
      register group_t loaded = ((group_t *)docs)[i * n_docs + doc_id];  // tid
      register uint16_t *doc_segment = (uint16_t *)(&loaded);
      for (auto j = 0; j < GROUP_SIZE; j++) {
        if (doc_segment[j] == 0) {
          no_more_load = true;
          break;
          // return;
        }
        int left = query_idx;
        int right = query_len - 1;
        int mid;
        while (left <= right) {
          mid = (left + right) >> 1;
          if (query_on_shm[mid] < doc_segment[j]) {
            left = mid + 1;
          } else {
            right = mid - 1;
          }
        }
        query_idx = left;  // update the query index

        if (query_idx < query_len) {
          tmp_score += (query_on_shm[query_idx] == doc_segment[j]);
        }
      }
      __syncwarp();
    }
    scores[doc_id] = tmp_score / max(query_len, doc_lens[doc_id]);  // tid
    d_index[doc_id] = doc_id;
  }
}
__global__ void pre_process_global(uint16_t *d_docs, int *d_doc_lens,int *docs, int n_docs,int cuda_docs_len,int ayer_1_offset,int layer_1_total_offset) {
  // 获取线程索引
  constexpr auto group_sz = sizeof(group_t) / sizeof(uint16_t);
  register auto layer_0_stride = n_docs * group_sz;

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  // 检查索引是否有效
  if (i < n_docs) {
  
    for (int j = 0; j < cuda_docs_len; j++) {
      auto layer_0_offset = j / group_sz;

      auto layer_2_offset = j % group_sz;
      auto final_offset = layer_0_offset * layer_0_stride +
                          layer_1_total_offset + layer_2_offset;
      d_docs[final_offset] = docs[j];
    }
  }
  d_doc_lens[i] = cuda_docs_len;
}

void pre_process(std::vector<std::vector<uint16_t>> &docs, uint16_t *h_docs,
                 std::vector<int> &h_doc_lens_vec) {
  auto n_docs = docs.size();

  constexpr auto group_sz = sizeof(group_t) / sizeof(uint16_t);
  auto layer_0_stride = n_docs * group_sz;
  constexpr auto layer_1_stride = group_sz;
auto numProcs = omp_get_num_procs() ;

omp_set_num_threads(8);
#pragma omp parallel
{
#pragma omp for 
  for (int i = 0; i < docs.size(); i++) {
    auto layer_1_offset = i;
    auto layer_1_total_offset = layer_1_offset * layer_1_stride;
    for (int j = 0; j < docs[i].size(); j++) {
      auto layer_0_offset = j / group_sz;

      auto layer_2_offset = j % group_sz;
      auto final_offset = layer_0_offset * layer_0_stride +
                          layer_1_total_offset + layer_2_offset;
      h_docs[final_offset] = docs[i][j];
    }
    h_doc_lens_vec[i] = docs[i].size();
  }
  }
}

  void doc_query_scoring_gpu_function(
    std::vector<std::vector<uint16_t>> &querys,
    std::vector<std::vector<uint16_t>> &docs, std::vector<uint16_t> &lens,
    std::vector<std::vector<int>> &indices  // shape [querys.size(), TOPK]
) {
  auto n_docs = docs.size();
  uint16_t *d_docs = nullptr;
  int *d_doc_lens = nullptr;

  uint16_t *h_docs = new uint16_t[MAX_DOC_SIZE * n_docs];

  std::vector<int> h_doc_lens_vec(n_docs);

  std::thread t1(pre_process, std::ref(docs), h_docs, std::ref(h_doc_lens_vec));

  hipStream_t stream = hipStreamPerThread;
  hipMallocAsync(&d_docs, sizeof(uint16_t) * MAX_DOC_SIZE * n_docs, stream);
  hipMallocAsync(&d_doc_lens, sizeof(int) * n_docs, stream);

  hipDeviceProp_t device_props;
  hipGetDeviceProperties(&device_props, 0);

  hipSetDevice(0);

  int block = N_THREADS_IN_ONE_BLOCK;
  int grid = (n_docs + block - 1) / block;
  int querys_len = querys.size();


  hipStream_t *streams;
  streams = (hipStream_t *)malloc(querys_len * sizeof(hipStream_t));
  t1.join();
  hipMemcpyAsync(d_docs, h_docs, sizeof(uint16_t) * MAX_DOC_SIZE * n_docs,
                  hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(d_doc_lens, h_doc_lens_vec.data(), sizeof(int) * n_docs,
                  hipMemcpyHostToDevice, stream);
 
  for (int i = 0; i < querys_len; ++i) {
    // init indices
    nvtxRangePushA("Loop start");
    uint16_t *d_query = nullptr;
    float *d_scores = nullptr;
    int *s_indices= nullptr;

    nvtxRangePushA("stream create");
    hipStreamCreate(&streams[i]);
    nvtxRangePop();

    auto &query = querys[i];
    const size_t query_len = query.size();
    
    hipMallocAsync(&d_query, sizeof(uint16_t) * query_len, streams[i]);
    hipMemcpyAsync(d_query, query.data(), sizeof(uint16_t) * query_len,
                    hipMemcpyHostToDevice, streams[i]);
    hipMallocAsync(&d_scores, sizeof(float) * n_docs, streams[i]);
    hipMallocAsync(&s_indices, sizeof(int) * n_docs, streams[i]);

    docQueryScoringCoalescedMemoryAccessSampleKernel<<<grid, block, 0,
                                                       streams[i]>>>(
        d_docs, d_doc_lens, n_docs, d_query, query_len, d_scores, s_indices);


        nvtxRangePushA("thrust device_ptr");
        thrust::device_ptr<float> scores_key(d_scores);
        thrust::device_ptr<int> s_indices_value(s_indices);
        nvtxRangePop();

        nvtxRangePushA("sort_by_key");
        thrust::sort_by_key(scores_key, scores_key + n_docs, s_indices_value,thrust::greater<float>());
        nvtxRangePop();

        std::vector<int> host_indices_temp(TOPK); // why
        hipMemcpyAsync(host_indices_temp.data(), s_indices, sizeof(int) * TOPK, hipMemcpyDeviceToHost, streams[i]);
        hipFreeAsync(s_indices, streams[i]);
        hipFreeAsync(d_scores, streams[i]);
        hipFreeAsync(d_query, streams[i]);
        indices.push_back(host_indices_temp);
        nvtxRangePop();
      
    }

  // deallocation
  // hipFree(d_docs);
  // hipFreeAsync(d_query);
  // hipFree(d_scores);
  // hipFree(d_doc_lens);
  // free(h_docs);
}
