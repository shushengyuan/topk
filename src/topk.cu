#include "hip/hip_runtime.h"
#include <nvtx3/nvToolsExt.h>
// #include <omp.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>

#include <thread>

#include "assert.h"
#include "topk.h"

typedef uint4 group_t;  // uint32_t
#define CHECK(res)          \
  if (res != hipSuccess) { \
    exit(-1);               \
  }

// device A100
//  cpu sort :
//  yuan trust sort L: 3002 ms
//  yuan trust sort L: 2750 ms

void __global__ docQueryScoringCoalescedMemoryAccessSampleKernel(
    const __restrict__ uint16_t *docs, const uint16_t *doc_lens,
    const size_t n_docs, const uint16_t *d_query, const int bach_now,
    const uint16_t *query_lens_d, float *scores, int *s_indices,
    const uint32_t *d_query_sum) {
  // each thread process one doc-query pair scoring task
  register auto tidx = blockIdx.x * blockDim.x + threadIdx.x,
                tnumx = gridDim.x * blockDim.x;

  if (tidx >= n_docs * bach_now) {
    return;
  }
  __shared__ uint32_t query_on_shm[MAX_QUERY_SIZE];
  // __shared__ uint32_t doc_lens_on_shm[n_docs];

  for (auto doc_id = tidx; doc_id < n_docs * bach_now; doc_id += tnumx) {
    register int query_idx = 0;

    register float tmp_score = 0.;

    register bool no_more_load = false;
    register int q_id = doc_id / n_docs;
    register auto query_len = query_lens_d[q_id];
    register auto start_index = d_query_sum[q_id] - d_query_sum[0];
    register auto doc_index = doc_id % n_docs;

#pragma unroll
    for (auto i = threadIdx.y; i < query_len; i += blockDim.y) {
      query_on_shm[i] =
          d_query[start_index + i];  // not very efficient query loading
      // temporally, as assuming its not
      // hotspot
    }
    __syncthreads();

    for (auto i = 0; i < MAX_DOC_SIZE / GROUP_SIZE; i++) {
      if (no_more_load) {
        break;
      }
      register group_t loaded =
          ((group_t *)docs)[i * n_docs + doc_index];  // tidx
      register uint16_t *doc_segment = (uint16_t *)(&loaded);
      for (auto j = 0; j < GROUP_SIZE; j++) {
        if (doc_segment[j] == 0) {
          no_more_load = true;
          break;
          // return;
        }
        while (query_idx < query_len &&
               query_on_shm[query_idx] < doc_segment[j]) {
          ++query_idx;
        }

        if (query_idx < query_len) {
          tmp_score += (query_on_shm[query_idx] == doc_segment[j]);
        }
      }
      __syncwarp();
    }

    scores[doc_id] = tmp_score / max(query_len, doc_lens[doc_index]);  // tidx
    s_indices[doc_id] = doc_index;
  }
}
__global__ void pre_process_global(const uint16_t *temp_docs, uint16_t *d_docs,
                                   const uint16_t *d_doc_lens,
                                   const size_t n_docs,
                                   const uint32_t *d_doc_sum) {
  register auto group_sz = sizeof(group_t) / sizeof(uint16_t);
  register auto layer_0_stride = n_docs * group_sz;
  register auto layer_1_stride = group_sz;

  register auto tidx = blockIdx.x * blockDim.x + threadIdx.x,
                tnumx = gridDim.x * blockDim.x;
  register auto tidy = blockIdx.y * blockDim.y + threadIdx.y,
                tnumy = gridDim.y * blockDim.y;
  // #pragma unroll
  for (auto i = tidx; i < n_docs; i += tnumx) {
    register auto layer_1_offset = i;
    register auto layer_1_total_offset = layer_1_offset * layer_1_stride;
    // #pragma unroll
    for (auto j = tidy; j < d_doc_lens[i]; j += tnumy) {
      register auto layer_0_offset = j / group_sz;
      register auto layer_2_offset = j % group_sz;
      register auto final_offset = layer_0_offset * layer_0_stride +
                                   layer_1_total_offset + layer_2_offset;
      d_docs[final_offset] = temp_docs[d_doc_sum[i] + j];
    }
  }
}

void query_thread(std::vector<std::vector<uint16_t>> &querys, uint16_t *h_query,
                  uint32_t *query_lens_sum) {
#pragma unroll
  for (size_t i = 0; i < querys.size(); i++) {
    auto querys_size = querys[i].size();

#pragma unroll
    for (size_t j = 0; j < querys_size; j++) {
      h_query[query_lens_sum[i] + j] = querys[i][j];
    }
  }
}

void pre_process(std::vector<std::vector<uint16_t>> &docs, uint16_t *h_docs,
                 uint32_t *h_docs_vec, size_t start_idx, size_t lens) {
#pragma unroll
  for (size_t i = start_idx; i < lens; i++) {
    auto doc_size = docs[i].size();
#pragma unroll
    for (size_t j = 0; j < doc_size; j++) {
      h_docs[h_docs_vec[i] + j] = docs[i][j];
    }
  }
}

void doc_query_scoring_gpu_function(
    std::vector<std::vector<uint16_t>> &querys,
    std::vector<std::vector<uint16_t>> &docs, std::vector<uint16_t> &lens,
    std::vector<std::vector<int>> &indices  // shape [querys.size(), TOPK]
) {
  // std::chrono::high_resolution_clock::time_point t1 =
  //     std::chrono::high_resolution_clock::now();

  size_t n_docs = docs.size();
  int total_querys_len = querys.size();
  const int BATCH_SIZE = total_querys_len;

  int block = N_THREADS_IN_ONE_BLOCK;
  int grid = ((BATCH_SIZE * n_docs) + block - 1) / block;

  uint16_t *d_docs = nullptr;
  uint16_t *d_doc_lens = nullptr;
  uint16_t *temp_docs = nullptr;
  std::vector<uint16_t> query_lens_vec(total_querys_len);
  std::vector<std::vector<int>> indices_pre(querys.size(),
                                            std::vector<int>(TOPK));

  uint32_t *h_docs_vec = new uint32_t[n_docs + 1];
  uint32_t *query_lens_sum = new uint32_t[total_querys_len + 1];
  h_docs_vec[0] = 0;
  query_lens_sum[0] = 0;
#pragma unroll_completely
  for (size_t i = 0; i < n_docs; i++) {
    h_docs_vec[i + 1] = h_docs_vec[i] + lens[i];
  }
#pragma unroll_completely
  for (size_t i = 0; i < total_querys_len; i++) {
    query_lens_vec[i] = querys[i].size();
    query_lens_sum[i + 1] = query_lens_sum[i] + querys[i].size();
  }

  uint16_t *h_docs = new uint16_t[h_docs_vec[n_docs]];
  uint16_t *h_query = new uint16_t[query_lens_sum[total_querys_len]];

  std::thread t_pre_1(pre_process, std::ref(docs), h_docs, h_docs_vec, 0,
                      n_docs / 2);
  std::thread t_pre_2(pre_process, std::ref(docs), h_docs, h_docs_vec,
                      n_docs / 2, n_docs);

  std::thread t_query(query_thread, std::ref(querys), h_query, query_lens_sum);

  hipDeviceProp_t device_props;
  hipGetDeviceProperties(&device_props, 0);
  hipSetDevice(0);

  dim3 numBlocks(32, 32);
  dim3 threadsPerBlock(32, 32);

  hipStream_t *streams;
  streams = (hipStream_t *)malloc(total_querys_len * sizeof(hipStream_t));
  for (int i = 0; i < total_querys_len; i++) {
    hipStreamCreate(&streams[i]);
  }

  hipMallocAsync(&d_docs, sizeof(uint16_t) * MAX_DOC_SIZE * n_docs,
                  streams[0]);
  uint32_t *d_doc_sum = nullptr;
  uint32_t *d_query_sum = nullptr;
  hipMallocAsync(&d_doc_sum, sizeof(uint32_t) * (n_docs + 1), streams[0]);
  hipMemcpyAsync(d_doc_sum, h_docs_vec, sizeof(uint32_t) * (n_docs + 1),
                  hipMemcpyHostToDevice, streams[0]);
  hipMallocAsync(&d_query_sum, sizeof(uint32_t) * (total_querys_len + 1),
                  streams[1]);
  hipMemcpyAsync(d_query_sum, query_lens_sum,
                  sizeof(uint32_t) * (total_querys_len + 1),
                  hipMemcpyHostToDevice, streams[1]);
  hipMallocAsync(&temp_docs, sizeof(uint16_t) * h_docs_vec[n_docs],
                  streams[2]);
  hipMallocAsync(&d_doc_lens, sizeof(uint16_t) * n_docs, streams[1]);
  hipMemcpyAsync(d_doc_lens, lens.data(), sizeof(uint16_t) * n_docs,
                  hipMemcpyHostToDevice, streams[1]);
  t_pre_1.join();
  t_pre_2.join();
  hipMemcpyAsync(temp_docs, h_docs, sizeof(uint16_t) * h_docs_vec[n_docs],
                  hipMemcpyHostToDevice, streams[2]);

  hipStreamSynchronize(streams[1]);
  hipStreamSynchronize(streams[2]);
  // nvtxRangePushA("pre_process_global start");
  pre_process_global<<<numBlocks, threadsPerBlock, 0, streams[0]>>>(
      temp_docs, d_docs, d_doc_lens, n_docs, d_doc_sum);
  // hipFreeAsync(&d_doc_sum, streams[0]);
  // hipFreeAsync(&temp_docs, streams[1]);
  // nvtxRangePop();

  // std::chrono::high_resolution_clock::time_point t2 =
  //     std::chrono::high_resolution_clock::now();

  // std::cout
  //     << "init cost "
  //     << std::chrono::duration_cast<std::chrono::milliseconds>(t2 -
  //     t1).count()
  // free(h_docs);

  uint16_t *query_lens_d = nullptr;
  hipMallocAsync(&query_lens_d, sizeof(uint16_t) * total_querys_len,
                  streams[1]);
  hipMemcpyAsync(query_lens_d, query_lens_vec.data(),
                  sizeof(uint16_t) * total_querys_len, hipMemcpyHostToDevice,
                  streams[1]);
  hipStreamSynchronize(streams[0]);
  hipStreamSynchronize(streams[1]);
  t_query.join();
  auto i = 0;

  while (i < total_querys_len) {
    auto bach_now = BATCH_SIZE;
    if (i + BATCH_SIZE == total_querys_len) {
      bach_now = BATCH_SIZE;
      // break;
    } else if (i + BATCH_SIZE > total_querys_len) {
      bach_now = total_querys_len - i;
    }
    if (bach_now == 0) {
      break;
    }

    // nvtxRangePushA("query start");

    float *d_scores = nullptr;
    int *s_indices = nullptr;
    uint16_t *d_query = nullptr;

    hipMallocAsync(
        &d_query,
        sizeof(uint16_t) * (query_lens_sum[i + bach_now] - query_lens_sum[i]),
        streams[i]);

    hipMemcpyAsync(
        d_query, h_query + query_lens_sum[i],
        sizeof(uint16_t) * (query_lens_sum[i + bach_now] - query_lens_sum[i]),
        hipMemcpyHostToDevice, streams[i]);

    hipMallocAsync(&d_scores, sizeof(float) * n_docs * bach_now, streams[i]);
    hipMallocAsync(&s_indices, sizeof(int) * n_docs * bach_now, streams[i]);
    docQueryScoringCoalescedMemoryAccessSampleKernel<<<grid, block, 0,
                                                       streams[i]>>>(
        d_docs, d_doc_lens, n_docs, d_query, bach_now, query_lens_d + i,
        d_scores, s_indices, d_query_sum + i);
    // free(h_query);
    // nvtxRangePop();
    void *d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;
    int *d_sort_index = nullptr;
    float *d_sort_scores = nullptr;

    hipMallocAsync(&d_sort_index, sizeof(int) * n_docs, streams[i]);
    hipMallocAsync(&d_sort_scores, sizeof(float) * n_docs, streams[i]);

    hipcub::DeviceRadixSort::SortPairsDescending(
        d_temp_storage, temp_storage_bytes, d_scores, d_sort_scores, s_indices,
        d_sort_index, n_docs);
    // Allocate temporary storage
    hipMallocAsync(&d_temp_storage, temp_storage_bytes, streams[i]);
    // hipDeviceSynchronize();
#pragma unroll
    for (int j = 0; j < bach_now; ++j) {
      // nvtxRangePushA("sort_by_key");
      hipcub::DeviceRadixSort::SortPairsDescending(
          d_temp_storage, temp_storage_bytes, d_scores + j * n_docs,
          d_sort_scores, s_indices + j * n_docs, d_sort_index, n_docs);

      hipMemcpyAsync(indices_pre[j + i].data(), d_sort_index,
                      sizeof(int) * TOPK, hipMemcpyDeviceToHost, streams[j]);
      hipMemsetAsync(d_sort_index, 0, n_docs * sizeof(int), streams[j]);
      hipMemsetAsync(d_sort_scores, 0, n_docs * sizeof(float), streams[j]);
      // hipFree(d_temp_storage);
      // nvtxRangePop();
    }
    // hipFreeAsync(s_indices, streams[i]);
    // hipFreeAsync(d_scores, streams[i]);
    // hipFreeAsync(d_query, streams[i]);

    i += bach_now;
  }

  indices = indices_pre;
  // hipFreeAsync(s_indices, streams[0]);
  // hipFreeAsync(d_scores, streams[0]);
  // hipFreeAsync(d_query, streams[0]);
  // // deallocation
  // hipFree(d_docs);
  // hipFree(d_scores);
  // hipFreeAsync(d_docs, streams[0]);
  // hipFreeAsync(d_doc_lens, streams[0]);
  // hipFree(d_doc_lens);
}